#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <vector>
#include <fstream>


#define V 333
#define E 800
#define MAX_WEIGHT 1000000
#define TRUE    1
#define FALSE   0

typedef int boolean;

struct Edge {
    int u;
    int v;
};

struct Vertex {
    int title;
    boolean visited;
};


__global__ void Find_Vertex(Vertex* vertices, Edge* edges, double* weights, double* length, double* updateLength, int* path) {
    int u = blockIdx.x * blockDim.x + threadIdx.x;
    if (u < V && vertices[u].visited == FALSE) {
        vertices[u].visited = TRUE;

        int uTitle = vertices[u].title;
        int lengthU = length[u];

        for (int i = 0; i < E; i++) {
            if (edges[i].u == uTitle) {
                int v = edges[i].v;
                double weight = weights[i];

                if (weight < MAX_WEIGHT) {
                    double potentialLength = lengthU + weight;

                    if (updateLength[v] > potentialLength) {
                        updateLength[v] = potentialLength;
                        int k = 0;
                        while (path[u * V + k] != MAX_WEIGHT)
                        {
                            path[v * V + k] = path[u * V + k];
                            k++;
                        }
                        path[v * V + k] = u;
                        path[v * V + k + 1] = MAX_WEIGHT;
                    }
                }
            }
        }
    }
}

__global__ void Update_Paths(Vertex* vertices, double* length, double* updateLength)
{
    int u = blockIdx.x * blockDim.x + threadIdx.x;

    if (u < V && length[u] > updateLength[u]) {
        length[u] = updateLength[u];
        vertices[u].visited = FALSE;
    }
    updateLength[u] = length[u];
}

void printArray(int* array) {
    for (int i = 0; i < V; i++) {
        std::cout << "Shortest Path to Vertex " << i << " is " << array[i] << std::endl;
    }
}
int konvertuj(std::string parametar)
{
    int broj = 0;
    for (int i = 0; parametar[i] != '\0'; i++)
    {
        broj = broj * 10 + parametar[i] - 48;
    }
    return broj;
}
std::vector<int> flattenVector(const std::vector<std::vector<int>>& input) {
    std::vector<int> result;
    for (const auto& innerVector : input) {
        result.insert(result.end(), innerVector.begin(), innerVector.end());
    }
    return result;
}

int main(int brojParametara, char* parametri[]) {
    if (brojParametara < 3)
        return 0;

    int firstNode = konvertuj(parametri[1]);
    int lastNode = konvertuj(parametri[2]);
    if (firstNode > lastNode) {
        int temp = firstNode;
        firstNode = lastNode;
        lastNode = temp;
    }
    std::vector<Vertex> vertices(V);
    std::vector<Edge> edges(E);
    std::vector<double> weights(E);
    std::vector<double> len(V, MAX_WEIGHT);
    std::vector<double> updateLength(V, MAX_WEIGHT);
    std::vector<int> path((V * V), MAX_WEIGHT);


    hipEvent_t timeStart, timeEnd;
    float runningTime;

    hipEventCreate(&timeStart);
    hipEventCreate(&timeEnd);



    for (int i = 0; i < V; ++i) {
        Vertex a = { i, FALSE };
        vertices[i] = a;
    }

    std::ifstream mapa("mapa.txt");
    if (!mapa.is_open())
    {
        std::cout << "Nemoguce otvoriti mapu";
        return 0;
    }
    double c;
    int a, b;
    int broj = 0;
    while (mapa >> a >> b >> c) {
        Edge e = { a, b };
        edges[broj] = e;
        weights[broj] = c;
        broj++;
    }


    Vertex* d_V;
    Edge* d_E;
    double* d_W;
    double* d_L;
    double* d_C;
    int* d_P;

    hipMalloc((void**)&d_P, sizeof(int) * V * V);

    //hipMalloc((void**)&d_P, sizeof(int) * V);
    hipMalloc((void**)&d_V, sizeof(Vertex) * V);
    hipMalloc((void**)&d_E, sizeof(Edge) * E);
    hipMalloc((void**)&d_W, sizeof(double) * E);
    hipMalloc((void**)&d_L, sizeof(double) * V);
    hipMalloc((void**)&d_C, sizeof(double) * V);
    //hipMemcpy(path.data(), d_P, sizeof(int) * V * V, hipMemcpyDeviceToHost);

    hipMemcpy(d_P, path.data(), sizeof(int) * V * V, hipMemcpyHostToDevice);
    //hipMemcpy(d_P, flattenedPath.data(), sizeof(int) * flattenedPath.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_V, vertices.data(), sizeof(Vertex) * V, hipMemcpyHostToDevice);
    hipMemcpy(d_E, edges.data(), sizeof(Edge) * E, hipMemcpyHostToDevice);
    hipMemcpy(d_W, weights.data(), sizeof(double) * E, hipMemcpyHostToDevice);
    hipMemcpy(d_L, len.data(), sizeof(double) * V, hipMemcpyHostToDevice);
    hipMemcpy(d_C, updateLength.data(), sizeof(double) * V, hipMemcpyHostToDevice);

    Vertex root = { firstNode, FALSE };
    root.visited = TRUE;

    len[root.title] = 0;
    updateLength[root.title] = 0;
    hipMemcpy(d_L, len.data(), sizeof(int) * V, hipMemcpyHostToDevice);
    hipMemcpy(d_C, updateLength.data(), sizeof(int) * V, hipMemcpyHostToDevice);

    hipEventRecord(timeStart, 0);

    for (int i = 0; i < V; i++) {

        Find_Vertex << <(V + 255) / 256, 256 >> > (d_V, d_E, d_W, d_L, d_C, d_P);
        hipDeviceSynchronize();
        Update_Paths << <(V + 255) / 256, 256 >> > (d_V, d_L, d_C);
        hipDeviceSynchronize();

    }

    hipEventRecord(timeEnd, 0);
    hipEventSynchronize(timeEnd);
    hipEventElapsedTime(&runningTime, timeStart, timeEnd);

    hipMemcpy(len.data(), d_L, sizeof(int) * V, hipMemcpyDeviceToHost);


    hipMemcpy(path.data(), d_P, sizeof(int) * V * V, hipMemcpyDeviceToHost);

    for (int i = 0; i < V; i++)
    {
        int j = 0;
        if (i == lastNode)
        {
            std::cout << "[";
            while (i == lastNode && path[i * V + j] != MAX_WEIGHT)
            {
                std::cout << path[i * V + j] << ",";
                j++;
            }
            std::cout << lastNode << "]";
        }
    }

    hipFree(d_V);
    hipFree(d_E);
    hipFree(d_W);
    hipFree(d_L);
    hipFree(d_C);
    hipFree(d_P);
    hipEventDestroy(timeStart);
    hipEventDestroy(timeEnd);

    return 0;
}